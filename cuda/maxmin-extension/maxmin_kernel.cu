#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <stdio.h>

template <typename scalar_t>
__global__ void maxmin_cuda_forward_kernel(
    const scalar_t* __restrict__ input,
    size_t outer_size,
    size_t axis_length,
    size_t inner_stride,
    int32_t group_size,
    scalar_t* __restrict__ argsort,
    scalar_t* __restrict__ output) {
  const int outer_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int axis_idx = blockIdx.y * blockDim.y + threadIdx.y;
  const int stride_idx = blockIdx.z * blockDim.z + threadIdx.z;
  const int inner_idx = group_size * axis_idx;
  if (outer_idx < outer_size && stride_idx < inner_stride) {
    
    const int start_idx = inner_stride * axis_length * outer_idx + inner_idx * inner_stride + stride_idx;
    
    if (inner_idx < axis_length - 1) { // TODO: FIX CONDITION FOR WHAT HAPPENS AT ENDS OF AXIS (maybe just fix input to always be a multiple?)

      // copy input to output and modify output in-place
      for (int i = 0; i < group_size; i++) {
        output[start_idx + inner_stride * i] = input[start_idx + inner_stride * i];
        argsort[start_idx + inner_stride * i] = i;
      }

      // insertion sort
      for (int i = 1; i < group_size; i++) {
        scalar_t key = output[start_idx + inner_stride * i];
        int j = i - 1;
        while (j >= 0 && output[start_idx + inner_stride * j] > key) {
          output[start_idx + inner_stride * (j + 1)] = output[start_idx + inner_stride * j];
          argsort[start_idx + inner_stride * (j + 1)] = argsort[start_idx + inner_stride * j];
          j = j - 1;
        }
        output[start_idx + inner_stride * (j + 1)] = key;
        argsort[start_idx + inner_stride * (j + 1)] = i;
      }

    } else if (inner_idx < axis_length) {
      // In range, but at end of sorting axis
      output[start_idx] = input[start_idx];
    }

  }

/*
  // REGULAR C++ INSERTION SORT CODE
  int i, key, j;
  for (i = 1; i < n; i++) {
      key = arr[i];
      j = i - 1;

        // Move elements of arr[0..i-1], that are
        // greater than key, to one position ahead
        // of their current position 
      while (j >= 0 && arr[j] > key) {
          arr[j + 1] = arr[j];
          j = j - 1;
      }
      arr[j + 1] = key;
  }
*/

 


}

std::vector<at::Tensor> maxmin_cuda_forward(
    at::Tensor input,
    int32_t axis, 
    int32_t group_size) {
  const auto num_dims = input.ndimension();
  const auto axis_length = input.size(axis);
  const int true_axis = (axis == -1) ? num_dims - 1 : axis;

  int outer_size = 1;
  for (int i = 0; i < true_axis; ++i) {
    outer_size *= input.size(i);
  };
  int inner_stride = 1;
  for (int i = true_axis + 1; i < num_dims; i++) {
    inner_stride *= input.size(i);
  }

  dim3 block(8, 8, 8);
  dim3 grid((outer_size + 7) / 8, (axis_length + 15) / 16, (inner_stride + 7) / 8);

  auto output = at::zeros_like(input);
  auto argsort = at::zeros_like(input);
  AT_DISPATCH_ALL_TYPES(input.type(), "maxmin_forward_cuda", ([&] {
    maxmin_cuda_forward_kernel<scalar_t><<<grid, block>>>(
        input.data<scalar_t>(),
        outer_size,
        axis_length,
        inner_stride,
        group_size,
        argsort.data<scalar_t>(),
        output.data<scalar_t>());
  }));
  std::vector<at::Tensor> outputs;
  outputs.push_back(output);
  outputs.push_back(argsort);
  return outputs;
}



template <typename scalar_t>
__global__ void maxmin_cuda_backward_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ grad,
    const scalar_t* __restrict__ argsort,
    size_t outer_size,
    size_t axis_length,
    size_t inner_stride,
    int32_t group_size,
    scalar_t* __restrict__ output_grad) {
  const int outer_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int axis_idx = blockIdx.y * blockDim.y + threadIdx.y;
  const int stride_idx = blockIdx.z * blockDim.z + threadIdx.z;
  const int inner_idx = group_size * axis_idx;

  if (outer_idx < outer_size && stride_idx < inner_stride) {
    const int start_idx = inner_stride * axis_length * outer_idx + inner_idx * inner_stride + stride_idx;
    if (inner_idx < axis_length - 1) { 

      // use sorted indices to undo the sort on the gradient
      for (int i = 0; i < group_size; i++) {
        int original_i = argsort[start_idx + inner_stride * i];
        output_grad[start_idx + inner_stride * original_i] = grad[start_idx + inner_stride * i];
      }

      
    } else if (inner_idx < axis_length) {
      // In range, but at end of sorting axis
      output_grad[start_idx] = grad[start_idx];
    }

  }

}

at::Tensor maxmin_cuda_backward(
    at::Tensor input,
    at::Tensor grad,
    int32_t axis,
    int32_t group_size,
    at::Tensor argsort) {
  const auto num_dims = input.ndimension();
  const auto axis_length = input.size(axis);
  const int true_axis = (axis == -1) ? num_dims - 1 : axis;

  int outer_size = 1;
  for (int i = 0; i < true_axis; ++i) {
    outer_size *= input.size(i);
  };
  int inner_stride = 1;
  for (int i = true_axis + 1; i < num_dims; i++) {
    inner_stride *= input.size(i);
  }

  dim3 block(8, 8, 8);
  dim3 grid((outer_size + 7) / 8, (axis_length + 15) / 16, (inner_stride + 7) / 8);

  auto output_grad = at::zeros_like(grad);

  AT_DISPATCH_ALL_TYPES(input.type(), "maxmin_backward_cuda", ([&] {
    maxmin_cuda_backward_kernel<scalar_t><<<grid, block>>>(
        input.data<scalar_t>(),
        grad.data<scalar_t>(),
        argsort.data<scalar_t>(),
        outer_size,
        axis_length,
        inner_stride,
        group_size,
        output_grad.data<scalar_t>());
  }));
  return output_grad;
}

